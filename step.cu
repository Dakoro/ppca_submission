#include <cstdlib>
#include <iostream>
#include <cmath>
#include <iostream>
#include <chrono>
#include <cstdlib>
#include <ctime>
#include <cfloat>
#include <random>
#include <hip/hip_runtime.h>
#include <omp.h>

static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b) {
    return (a + b - 1)/b;
}

static inline int roundup(int a, int b) {
    return divup(a, b) * b;
}

__global__ void mykernel(float* r, const float* d, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= n || j >= n)
        return;
    float v = HUGE_VALF;
    for (int k = 0; k < n; ++k) {
        float x = d[n*i + k];
        float y = d[n*k + j];
        float z = x + y;
        v = min(v, z);
    }
    r[n*i + j] = v;
}

__global__ void mykernel_v2(float* r, const float* d, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= n || j >= n)
        return;
    float v = HUGE_VALF;
    for (int k = 0; k < n; ++k) {
        float x = d[n*j + k];
        float y = d[n*k + i];
        float z = x + y;
        v = min(v, z);
    }
    r[n*j + i] = v;
}

__global__ void mykernel_v3(float* r, const float* d, int n, int nn) {
    int ia = threadIdx.x;
    int ja = threadIdx.y;
    int ic = blockIdx.x;
    int jc = blockIdx.y;

    const float* t = d + nn * nn;

    float v[8][8];
    for (int ib = 0; ib < 8; ++ib) {
        for (int jb = 0; jb < 8; ++jb) {
            v[ib][jb] = HUGE_VALF;
        }
    }
    for (int k = 0; k < n; ++k) {
        float x[8];
        float y[8];
        for (int ib = 0; ib < 8; ++ib) {
            int i = ic * 64 + ib * 8 + ia;
            x[ib] = t[nn*k + i];
        }
        for (int jb = 0; jb < 8; ++jb) {
            int j = jc * 64 + jb * 8 + ja;
            y[jb] = d[nn*k + j];
        }
        for (int ib = 0; ib < 8; ++ib) {
            for (int jb = 0; jb < 8; ++jb) {
                v[ib][jb] = min(v[ib][jb], x[ib] + y[jb]);
            }
        }
    }
    for (int ib = 0; ib < 8; ++ib) {
        for (int jb = 0; jb < 8; ++jb) {
            int i = ic * 64 + ib * 8 + ia;
            int j = jc * 64 + jb * 8 + ja;
            if (i < n && j < n) {
                r[n*i + j] = v[ib][jb];
            }
        }
    }
}
__global__ void myppkernel(const float* r, float* d, int n, int nn) {
    int ja = threadIdx.x;
    int i = blockIdx.y;

    float* t = d + nn * nn;

    for (int jb = 0; jb < nn; jb += 64) {
        int j = jb + ja;
        float v = (i < n && j < n) ? r[n*i + j] : HUGE_VALF;
        d[nn*i + j] = v;
        t[nn*j + i] = v;
    }
}

__global__ void mykernel_v4(float* r, const float* d, int n, int nn) {
    int ia = threadIdx.x;
    int ja = threadIdx.y;
    int ic = blockIdx.x;
    int jc = blockIdx.y;

    const float* t = d + nn * nn;

    __shared__ float xx[4][64];
    __shared__ float yy[4][64];

    float v[8][8];
    for (int ib = 0; ib < 8; ++ib) {
        for (int jb = 0; jb < 8; ++jb) {
            v[ib][jb] = HUGE_VALF;
        }
    }
    for (int ks = 0; ks < n; ks += 4) {
        int ija = ja * 8 + ia;
        int i = ic * 64 + ija;
        int j = jc * 64 + ija;
        for (int f = 0; f < 4; ++f) {
            int k = ks + f;
            xx[f][ija] = t[nn*k + i];
            yy[f][ija] = d[nn*k + j];
        }

        __syncthreads();

        #pragma unroll
        for (int f = 0; f < 4; ++f) {
            float y[8];
            for (int jb = 0; jb < 8; ++jb) {
                y[jb] = yy[f][jb * 8 + ja];
            }
            for (int ib = 0; ib < 8; ++ib) {
                float x = xx[f][ib * 8 + ia];
                for (int jb = 0; jb < 8; ++jb) {
                    v[ib][jb] = min(v[ib][jb], x + y[jb]);
                }
            }
        }

        __syncthreads();
    }
    for (int ib = 0; ib < 8; ++ib) {
        for (int jb = 0; jb < 8; ++jb) {
            int i = ic * 64 + ib * 8 + ia;
            int j = jc * 64 + jb * 8 + ja;
            if (i < n && j < n) {
                r[n*i + j] = v[ib][jb];
            }
        }
    }
}

void step_v2(float* r, const float* d, int n) {
    int nn = roundup(n, 64);

    // Allocate memory & copy data to GPU
    float* dGPU = NULL;
    CHECK(hipMalloc((void**)&dGPU, 2 * nn * nn * sizeof(float)));
    float* rGPU = NULL;
    CHECK(hipMalloc((void**)&rGPU, n * n * sizeof(float)));
    CHECK(hipMemcpy(rGPU, d, n * n * sizeof(float), hipMemcpyHostToDevice));

    // Run kernel
    {
        dim3 dimBlock(64, 1);
        dim3 dimGrid(1, nn);
        myppkernel<<<dimGrid, dimBlock>>>(rGPU, dGPU, n, nn);
        CHECK(hipGetLastError());
    }

    // Run kernel
    {
        dim3 dimBlock(8, 8);
        dim3 dimGrid(nn / 64, nn / 64);
        mykernel_v4<<<dimGrid, dimBlock>>>(rGPU, dGPU, n, nn);
        CHECK(hipGetLastError());
    }

    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(r, rGPU, n * n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}


void step(float* r, const float* d, int n) {
    // Allocate memory & copy data to GPU
    float* dGPU = NULL;
    CHECK(hipMalloc((void**)&dGPU, n * n * sizeof(float)));
    float* rGPU = NULL;
    CHECK(hipMalloc((void**)&rGPU, n * n * sizeof(float)));
    CHECK(hipMemcpy(dGPU, d, n * n * sizeof(float), hipMemcpyHostToDevice));

    // Run kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(n, dimBlock.x), divup(n, dimBlock.y));
    mykernel_v2<<<dimGrid, dimBlock>>>(rGPU, dGPU, n);
    CHECK(hipGetLastError());

    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(r, rGPU, n * n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}


int main() {
    constexpr int n = 4000;
    static float d[n*n];
    std::srand( ( unsigned int )std::time( nullptr ) );
    for (int i = 0; i < n*n; ++i) {
        std::random_device rd;  // Obtain a random number from hardware
        std::mt19937 gen(rd()); // Use Mersenne Twister engine
        std::uniform_real_distribution<> dis(0.0, 1.0); // Define the range
        float random_float = dis(gen);
        d[i] = random_float;
    }

    static float r[n*n];

    auto start = std::chrono::high_resolution_clock::now();
    step_v2(r, d, n);
    auto end = std::chrono::high_resolution_clock::now();
    auto ms_int = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    std::cout << ms_int.count() << " ms";
    // for (int i = 0; i < n; ++i) {
    //     for (int j = 0; j < n; ++j) {
    //         std::cout << r[i*n + j] << " ";
    //     }
    //     std::cout << "\n";
    // }
}