#include <hip/hip_runtime.h>
#include <iostream>

static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

// Single kernel to compute all correlations
__global__ void correlateKernel(int ny, int nx, const float* data, float* result) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (j <= i && i < ny) {  // Only compute upper triangle
        // Get pointers to the two rows
        const float* row_i = &data[i * nx];
        const float* row_j = &data[j * nx];
        
        // Step 1: Calculate means of both rows
        double mean_i = 0.0;
        double mean_j = 0.0;
        for (int k = 0; k < nx; k++) {
            mean_i += row_i[k];
            mean_j += row_j[k];
        }
        mean_i /= nx;
        mean_j /= nx;
        
        // Step 2: Calculate normalized dot product
        double numerator = 0.0;
        double sum_sq_i = 0.0;
        double sum_sq_j = 0.0;
        
        for (int k = 0; k < nx; k++) {
            double val_i = row_i[k] - mean_i;
            double val_j = row_j[k] - mean_j;
            
            numerator += val_i * val_j;
            sum_sq_i += val_i * val_i;
            sum_sq_j += val_j * val_j;
        }
        
        // Step 3: Compute correlation
        double denominator = sqrt(sum_sq_i * sum_sq_j);
        double correlation = 0.0;
        
        if (denominator > 1e-10) {
            correlation = numerator / denominator;
        }
        
        // Store result
        result[i + j * ny] = correlation;
    }
}

void correlate(int ny, int nx, const float* data, float* result) {
    // Allocate device memory
    float *d_data = NULL, *d_result = NULL;
    hipError_t err;
    
    // Define grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((ny + blockDim.x - 1) / blockDim.x, 
                 (ny + blockDim.y - 1) / blockDim.y);
                 
    // Allocate memory for input data
    CHECK(hipMalloc((void**)&d_data, ny * nx * sizeof(float)));
    
    // Allocate memory for result
    CHECK(hipMalloc((void**)&d_result, ny * ny * sizeof(float)));
    
    // Copy input data to device
    CHECK(hipMemcpy(d_data, data, ny * nx * sizeof(float), hipMemcpyHostToDevice));
    
    // Initialize result matrix with zeros to avoid uninitialized memory issues
    CHECK(hipMemset(d_result, 0, ny * ny * sizeof(float)));
    
    // Launch kernel
    correlateKernel<<<gridDim, blockDim>>>(ny, nx, d_data, d_result);
    
    // Check for kernel errors
    CHECK(hipGetLastError());
    
    // Synchronize to ensure kernel completion
    hipDeviceSynchronize();
    
    // Copy result back to host
    CHECK(hipMemcpy(result, d_result, ny * ny * sizeof(float), hipMemcpyDeviceToHost));

    // Free device memory
    if (d_data) hipFree(d_data);
    if (d_result) hipFree(d_result);
}